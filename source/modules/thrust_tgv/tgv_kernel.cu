#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/adjacent_difference.h>
#include <algorithm>

#include <stdio.h>

#include "ThrustImage.cuh"
#include "thrust_operators.cuh"

template<typename PixelVector>
void print(ThrustImage<PixelVector>* ThrustImage, std::string title)
{
    std::cout << "ThrustImage: " << title << std::endl;
    for(int z = 0; z < ThrustImage->depth; z++)
    {
        for(int y = 0; y < ThrustImage->height; y++)
        {
            for(int x = 0; x < ThrustImage->width; x++)
            {
                std::cout << ThrustImage->getPixel(x,y, z) << "\t";
            }
            std::cout << std::endl;
        }
    }
}

template<typename PixelVector>
__host__ __device__
ThrustImage<PixelVector>* filter(ThrustImage<PixelVector>* f,
                                 const Pixel lambda,
                                 const unsigned int iteration_count,
                                 const unsigned int paint_iteration_interval,
                                 std::function<void(uint iteration_index, uint iteration_count, ThrustImage<PixelVector>*)> iteration_finished_callback
                                 )
{
    typedef ThrustImage<PixelVector> ThrustThrustImage;


#if THRUST_HOST_SYSTEM == THRUST_HOST_SYSTEM_OMP
    omp_set_num_threads(NUMBER_OF_THREADS);
#endif

    printf("TVL2, lambda=%f \n", lambda);

    /*
  f = new ThrustThrustImage(3,3);
  f->setPixel(0, 0, 1);
  f->setPixel(1, 0, 3);
  f->setPixel(2, 0, 7);

  f->setPixel(0, 1, 15);
  f->setPixel(1, 1, 25);
  f->setPixel(2, 1, 42);

  f->setPixel(0, 2, 85);
  f->setPixel(1, 2, 166);
  f->setPixel(2, 2, 512);
*/


    const Pixel sqrt_8 = std::sqrt(8.0);
    Pixel tau = 1.0 / sqrt_8;
    Pixel sigma = tau;

    Pixel theta = 1.0; // will be used later

    ThrustThrustImage* u = f->clone();
    ThrustThrustImage* p_x_temp = u->clone_uninitialized(); // memory allocation
    ThrustThrustImage* p_y_temp = u->clone_uninitialized(); // memory allocation
    ThrustThrustImage* p_x = u->clone_initialized(0); // memory allocation
    ThrustThrustImage* p_y = u->clone_initialized(0); // memory allocation

    ThrustThrustImage* p_z_temp;
    ThrustThrustImage* p_z;
    if(u->depth > 1) {
        p_z_temp = u->clone_uninitialized(); // memory allocation
        p_z = u->clone_initialized(0); // memory allocation
    }


    ThrustThrustImage* divergence_p = u->clone_uninitialized(); // memory allocation
    ThrustThrustImage* u_bar = u->clone(); // memory allocation
    ThrustThrustImage* u_previous = u->clone_uninitialized(); // memory allocation

    ThrustThrustImage* p_magnitude = u->clone_uninitialized(); // memory allocation

    for(uint iteration_index = 0; iteration_index < iteration_count; iteration_index++)
    {
        /* matlab primal dual TVL2
       *
          u_old = u;

          % dual update
          p = p + sigma*nabla*u_bar;
          norm_p = sqrt(p(1:N).^2 + p(N+1:2*N).^2);
          p = p./max(1,[norm_p; norm_p]);

          % primal update
          u = u_old - tau * nabla_t * p;
          u = (u + tau * lambda .* f)/(1 + tau * lambda);

          % overrelaxation
          u_bar = u + theta*(u - u_old);
      *
      */


        // u_old = u;
        u_previous->set_pixel_data_of(u);

        //p = p + sigma*nabla*u_bar;
        u_bar->forward_difference_x(p_x_temp);
        u_bar->forward_difference_y(p_y_temp);

        if(u->depth > 1) {
            u_bar->forward_difference_z(p_z_temp);
        }

        thrust::transform(p_x_temp->pixel_rows.begin(), p_x_temp->pixel_rows.end(),
                          p_x->pixel_rows.begin(), p_x_temp->pixel_rows.begin(),
                          MultiplyByConstantAndAddOperation<Pixel>(sigma) );
        thrust::transform(p_y_temp->pixel_rows.begin(), p_y_temp->pixel_rows.end(),
                          p_y->pixel_rows.begin(), p_y_temp->pixel_rows.begin(),
                          MultiplyByConstantAndAddOperation<Pixel>(sigma) );

        if(u->depth > 1) {
            thrust::transform(p_z_temp->pixel_rows.begin(), p_z_temp->pixel_rows.end(),
                              p_z->pixel_rows.begin(), p_z_temp->pixel_rows.begin(),
                              MultiplyByConstantAndAddOperation<Pixel>(sigma) );
        }


        // norm_p = sqrt(p(1:N).^2 + p(N+1:2*N).^2);
        thrust::transform(p_x_temp->pixel_rows.begin(), p_x_temp->pixel_rows.end(),
                          p_x->pixel_rows.begin(),
                          SquareOperation<Pixel>() );
        thrust::transform(p_y_temp->pixel_rows.begin(), p_y_temp->pixel_rows.end(),
                          p_y->pixel_rows.begin(),
                          SquareOperation<Pixel>() );

        if(u->depth > 1) {
            thrust::transform(p_z_temp->pixel_rows.begin(), p_z_temp->pixel_rows.end(),
                              p_z->pixel_rows.begin(),
                              SquareOperation<Pixel>() );
        }

        thrust::transform(p_x->pixel_rows.begin(), p_x->pixel_rows.end(),
                          p_y->pixel_rows.begin(), p_magnitude->pixel_rows.begin(),
                          thrust::plus<Pixel>() );

        if(u->depth > 1) {
            thrust::transform(p_magnitude->pixel_rows.begin(), p_magnitude->pixel_rows.end(),
                              p_z->pixel_rows.begin(), p_magnitude->pixel_rows.begin(),
                              thrust::plus<Pixel>() );
        }

        thrust::transform(p_magnitude->pixel_rows.begin(), p_magnitude->pixel_rows.end(),
                          p_magnitude->pixel_rows.begin(),
                          SquareRootOperation<Pixel>() );

        // p = p./max(1,[norm_p; norm_p]);
        thrust::transform(p_magnitude->pixel_rows.begin(), p_magnitude->pixel_rows.end(),
                          p_magnitude->pixel_rows.begin(),
                          MaxOperation<Pixel>(1.0) );

        thrust::transform(p_x_temp->pixel_rows.begin(), p_x_temp->pixel_rows.end(),
                          p_magnitude->pixel_rows.begin(), p_x->pixel_rows.begin(),
                          thrust::divides<Pixel>() );
        thrust::transform(p_y_temp->pixel_rows.begin(), p_y_temp->pixel_rows.end(),
                          p_magnitude->pixel_rows.begin(), p_y->pixel_rows.begin(),
                          thrust::divides<Pixel>() );

        if(u->depth > 1) {
            thrust::transform(p_z_temp->pixel_rows.begin(), p_z_temp->pixel_rows.end(),
                              p_magnitude->pixel_rows.begin(), p_z->pixel_rows.begin(),
                              thrust::divides<Pixel>() );
        }

        // u = u_old - tau * nabla_t * p;
        ThrustThrustImage::divergence(p_x, p_y, p_z,
                                      p_x_temp, p_y_temp, p_z_temp,
                                      divergence_p);

        divergence_p->scale(-tau, divergence_p);
        u_previous->add(divergence_p, u);

        thrust::transform(u->pixel_rows.begin(), u->pixel_rows.end(),
                          f->pixel_rows.begin(), u->pixel_rows.begin(),
                          L1DataTermOperation<Pixel>(tau*lambda) );

        // u_bar = u + theta*(u - u_old);
        thrust::transform(u->pixel_rows.begin(), u->pixel_rows.end(),
                          u_previous->pixel_rows.begin(), u_bar->pixel_rows.begin(),
                          thrust::minus<Pixel>());

        thrust::transform(u_bar->pixel_rows.begin(), u_bar->pixel_rows.end(),
                          u->pixel_rows.begin(), u_bar->pixel_rows.begin(),
                          MultiplyByConstantAndAddOperation<Pixel>(theta) );

        printf("TVL2, iteration=%d / %d \n", iteration_index, iteration_count);

        if(iteration_finished_callback != nullptr &&
           paint_iteration_interval > 0 &&
           iteration_index % paint_iteration_interval == 0 )
            iteration_finished_callback(iteration_index, iteration_count, u);
    }

    delete p_x_temp;
    delete p_y_temp;
    delete p_x;
    delete p_y;

    if(u->depth > 1) {
        delete p_z_temp;
        delete p_z;
    }

    delete divergence_p;
    delete u_previous;
    delete u_bar;

    delete p_magnitude;

    return u;
}



ThrustImage<DevicePixelVector>* filterGPU(ThrustImage<DevicePixelVector>* f,
                                          const Pixel lambda, const unsigned int iteration_count,
                                          const unsigned int paint_iteration_interval,
                                          std::function<void(uint iteration_index, uint iteration_count, ThrustImage<DevicePixelVector>*)>
                                          iteration_finished_callback)
{
    return filter(f, lambda, iteration_count, paint_iteration_interval, iteration_finished_callback);
}

ThrustImage<HostPixelVector>* filterCPU(ThrustImage<HostPixelVector>* f,
                                        const Pixel lambda, const unsigned int iteration_count,
                                        const unsigned int paint_iteration_interval,
                                        std::function<void(uint iteration_index, uint iteration_count, ThrustImage<HostPixelVector>*)> iteration_finished_callback)
{
    return filter(f, lambda, iteration_count, paint_iteration_interval, iteration_finished_callback);
}





