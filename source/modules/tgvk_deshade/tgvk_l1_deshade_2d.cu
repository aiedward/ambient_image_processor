#include "hip/hip_runtime.h"
/*
    Ambient Image Processor - A tool to perform several imaging tasks
    
    Copyright (C) 2016 Josef Koller

    https://github.com/josefkoller/ambient_image_processor    
    
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "tgv3_l1_2d.cu"
#include "tgvk_common_2d.cu"

// k >= 3

template<typename Pixel>
Pixel* tgvk_l1_deshade_launch_2d(Pixel* f_host,
                      const uint width, const uint height,
                      const Pixel lambda,
                      const uint iteration_count,
                      const uint paint_iteration_interval,
                      DeshadeIterationCallback2D<Pixel> iteration_finished_callback,
                      const uint order,
                      const Pixel* alpha,
                      Pixel** v_x_host, Pixel**v_y_host)
{
    uint voxel_count;
    dim3 block_dimension;
    dim3 grid_dimension;
    dim3 grid_dimension_x;
    dim3 grid_dimension_y;

    tgv_launch_part1_2d<Pixel>(
                        width, height,
                        voxel_count,
                        block_dimension,
                        grid_dimension,
                        grid_dimension_x,
                        grid_dimension_y);

    Pixel* f, *u;
    Pixel* u_previous, *u_bar, *p_x, *p_y, *p_xx, *p_yy;

    tgv_launch_part2_2d<Pixel>(f_host,
                voxel_count,
                &f, &u,
                &u_previous, &u_bar, &p_x, &p_y, &p_xx, &p_yy);

    Pixel *v_x, *v_y;
    Pixel *v_bar_x, *v_bar_y;
    Pixel *v_previous_x, *v_previous_y;
    Pixel *q_x, *q_y, *q_xy;
    Pixel *q_x2, *q_y2, *q_xy2;
    Pixel * q_temp;

    tgv_launch_part22_2d<Pixel>(
                voxel_count,
                &v_previous_x, &v_previous_y,
                &v_bar_x, &v_bar_y,
                &v_x, &v_y,
                &q_x, &q_y,
                &q_xy,
                &q_x2, &q_y2,
                &q_xy2, &q_temp);

    // order 3 -> 1x w, w_bar, w_previous, r, r2
    // order 4 -> 2x w, w_bar, w_previous, r, r2
    // order 5 -> 3x w, w_bar, w_previous, r, r2

    int k_minus_2 = order - 2;
    Pixel *w_x[k_minus_2], *w_y[k_minus_2], *w_xy[k_minus_2];
    Pixel *w_bar_x[k_minus_2], *w_bar_y[k_minus_2], *w_bar_xy[k_minus_2];
    Pixel *w_previous_x[k_minus_2], *w_previous_y[k_minus_2], *w_previous_xy[k_minus_2];
    Pixel *r_x[k_minus_2], *r_y[k_minus_2], *r_xy[k_minus_2];
    Pixel *r2_x[k_minus_2], *r2_y[k_minus_2], *r2_xy[k_minus_2];

    for(int i = 0; i < order - 2; i++)
    {
        tgv3_launch_part23_2d<Pixel>(
                    voxel_count,
                    &w_x[i], &w_y[i],
                    &w_xy[i],
                    &w_bar_x[i], &w_bar_y[i],
                    &w_bar_xy[i],
                    &w_previous_x[i], &w_previous_y[i],
                    &w_previous_xy[i],
                    &r_x[i], &r_y[i],
                    &r_xy[i],
                    &r2_x[i], &r2_y[i],
                    &r2_xy[i]);
    }

    // algorithm variables..
    const Pixel tau = 1.0 / std::sqrt(20.0);
    const Pixel sigma = tau;
    const Pixel theta = 1;

    // algorithm begin
    zeroInit_2d<<<grid_dimension, block_dimension>>>(
                                                    p_x, p_y,
                                                    p_xx, p_yy,
                                                    voxel_count);
    zeroInit_2d<<<grid_dimension, block_dimension>>>(
                                                    v_x, v_y,
                                                    v_bar_x, v_bar_y,
                                                    voxel_count);
    zeroInit2_2d<<<grid_dimension, block_dimension>>>(
                                                    q_x, q_y,
                                                    q_xy,
                                                    voxel_count);

    for(int i = 0; i < order - 2; i++)
    {
        zeroInit2_2d<<<grid_dimension, block_dimension>>>(
                                                        w_x[i], w_y[i],
                                                        w_xy[i],
                                                        voxel_count);
        zeroInit2_2d<<<grid_dimension, block_dimension>>>(
                                                        w_bar_x[i], w_bar_y[i],
                                                        w_bar_xy[i],
                                                        voxel_count);
        zeroInit2_2d<<<grid_dimension, block_dimension>>>(
                                                        r_x[i], r_y[i],
                                                        r_xy[i],
                                                        voxel_count);
    }

    clone2<<<grid_dimension, block_dimension>>>(f, u, u_bar, voxel_count);
    cudaCheckError( hipDeviceSynchronize() );

    for(uint iteration_index = 0; iteration_index < iteration_count; iteration_index++)
    {
        tgv_launch_forward_differences_2d<Pixel>(u_bar,
                p_x, p_y,
                width, height,
                block_dimension,
                grid_dimension_x,
                grid_dimension_y);

        tgv_kernel_part22_2d<<<grid_dimension, block_dimension>>>( v_bar_x, v_bar_y,
                                                                p_x, p_y,
                                                                p_xx, p_yy,
                                                                sigma, alpha[0], u_previous, u,
                                                                width, height);
        cudaCheckError( hipDeviceSynchronize() );

        tgv_launch_backward_differences_2d<Pixel>(
                p_x, p_y,
                p_xx, p_yy,
                width, height,
                block_dimension,
                grid_dimension_x,
                grid_dimension_y);
        cudaCheckError( hipDeviceSynchronize() );

        tgv_kernel_part4_tgv2_l1_2d<<<grid_dimension, block_dimension>>>(
                                                                p_x, p_y,
                                                                tau, u, f,
                                                                lambda,
                                                                u_previous, theta, u_bar,
                                                                width, height);
        cudaCheckError( hipDeviceSynchronize() );

        // dual update q

        tgv_launch_gradient2_2d<Pixel>(
                v_bar_x, v_bar_y,
                q_x2,q_y2,
                q_xy2, q_temp,
                width, height,
                block_dimension,
                grid_dimension,
                grid_dimension_x,
                grid_dimension_y);

        tgv3_kernel_part5_2d<<<grid_dimension, block_dimension>>>(
                v_x, v_y,
                v_previous_x, v_previous_y,
                q_x2, q_y2,
                q_xy2,
                q_x, q_y,
                q_xy,

                w_bar_x[0], w_bar_y[0],
                w_bar_xy[0],

                sigma, alpha[1],
                width, height);
        cudaCheckError( hipDeviceSynchronize() );

        tgv_launch_divergence2_2d<Pixel>(
                q_x, q_y,
                q_xy,
                q_x2, q_y2,
                q_temp,
                width, height,
                block_dimension,
                grid_dimension,
                grid_dimension_x,
                grid_dimension_y);
        cudaCheckError( hipDeviceSynchronize() );


        // primal update v

        tgv_kernel_part6_2d<<<grid_dimension, block_dimension>>>(
                v_x, v_y,
                q_x2, q_y2,
                p_xx, p_yy,
                v_previous_x, v_previous_y,
                v_bar_x, v_bar_y,
                tau, theta,
                width, height);
        cudaCheckError( hipDeviceSynchronize() );

        // dual update r

        for(int i = 0; i < order - 2; i++)
        {
            if(i % 2 == 1)
            {
                tgv_launch_gradient3_2d<Pixel>(
                        w_bar_x[i], w_bar_y[i],
                        w_bar_xy[i],

                        r2_x[i], r2_y[i],
                        r2_xy[i],

                        q_temp,

                        width, height,
                        block_dimension,
                        grid_dimension,
                        grid_dimension_x,
                        grid_dimension_y);
            }
            else
            {
                tgv_launch_gradient3_backward_2d<Pixel>(
                        w_bar_x[i], w_bar_y[i],
                        w_bar_xy[i],

                        r2_x[i], r2_y[i],
                        r2_xy[i],

                        q_temp,

                        width, height,
                        block_dimension,
                        grid_dimension,
                        grid_dimension_x,
                        grid_dimension_y);

            }

            if(i == order - 3)
            {
                tgv3_kernel_part52_2d<<<grid_dimension, block_dimension>>>(
                        r2_x[i], r2_y[i],
                        r2_xy[i],

                        r_x[i], r_y[i],
                        r_xy[i],

                        w_x[i], w_y[i],
                        w_xy[i],

                        w_previous_x[i], w_previous_y[i],
                        w_previous_xy[i],

                        sigma, alpha[i + 2],
                        width, height);
            }
            else
            {
                tgvk_kernel_part5_2d<<<grid_dimension, block_dimension>>>(
                         r2_x[i], r2_y[i],
                         r2_xy[i],

                         r_x[i], r_y[i],
                         r_xy[i],

                         w_x[i], w_y[i],
                         w_xy[i],

                         w_previous_x[i], w_previous_y[i],
                         w_previous_xy[i],

                         w_bar_x[i + 1], w_bar_y[i + 1],   // minus primal of next order
                         w_bar_xy[i + 1],

                         sigma, alpha[i + 2],
                         width, height);
            }
            cudaCheckError( hipDeviceSynchronize() );

            if(i % 2 == 1)
            {
                tgv_launch_divergence3_2d<Pixel>(
                        r_x[i], r_y[i],
                        r_xy[i],

                        r2_x[i], r2_y[i],
                        r2_xy[i],

                        q_temp,

                        width, height,
                        block_dimension,
                        grid_dimension,
                        grid_dimension_x,
                        grid_dimension_y);
            }
            else
            {
                tgv_launch_divergence3_forward_2d<Pixel>(
                        r_x[i], r_y[i],
                        r_xy[i],

                        r2_x[i], r2_y[i],
                        r2_xy[i],

                        q_temp,

                        width, height,
                        block_dimension,
                        grid_dimension,
                        grid_dimension_x,
                        grid_dimension_y);
            }

            // primal update w

            Pixel* q_x_tilt = i == 0 ? q_x : r_x[i - 1]; // minus dual of previous order
            Pixel* q_y_tilt = i == 0 ? q_y : r_y[i - 1];
            Pixel* q_xy_tilt = i == 0 ? q_xy : r_xy[i - 1];

            tgv3_kernel_part62_2d<<<grid_dimension, block_dimension>>>(
                    r2_x[i], r2_y[i],
                    r2_xy[i],

                    q_x_tilt, q_y_tilt,
                    q_xy_tilt,

                    w_x[i], w_y[i],
                    w_xy[i],

                    w_previous_x[i], w_previous_y[i],
                    w_previous_xy[i],

                    w_bar_x[i], w_bar_y[i],
                    w_bar_xy[i],

                    tau, theta,
                    width, height);
            cudaCheckError( hipDeviceSynchronize() );
        }


        bool stop = tgv2_deshade_iteration_callback_2d(
                    iteration_index, iteration_count, paint_iteration_interval,
                    u, v_x, v_y,
                    iteration_finished_callback, voxel_count);
        if(stop)
            break;
    }

    Pixel* destination = new Pixel[voxel_count];
    tgv_launch_part3_2d<Pixel>(
                destination,
                voxel_count,
                u_previous, u_bar,
                p_x, p_y,
                p_xx, p_yy,
                f, u);
    // copy v from the device to the host memory...
    *v_x_host = new Pixel[voxel_count];
    *v_y_host = new Pixel[voxel_count];
    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMemcpy(*v_x_host, v_x, size, hipMemcpyDeviceToHost) );
    cudaCheckError( hipMemcpy(*v_y_host, v_y, size, hipMemcpyDeviceToHost) );
    cudaCheckError( hipDeviceSynchronize() );

    tgv_launch_part32_2d<Pixel>(
              v_bar_x, v_bar_y,
              v_previous_x, v_previous_y,
                v_x, v_y,
                q_x, q_y,
                q_xy,
              q_x2, q_y2,
              q_xy2, q_temp);

    for(int i = 0; i < order - 2; i++)
    {
        tgv3_launch_part33_2d<Pixel>(
                    w_x[i], w_y[i],
                    w_xy[i],
                    w_bar_x[i], w_bar_y[i],
                    w_bar_xy[i],
                    w_previous_x[i], w_previous_y[i],
                    w_previous_xy[i],
                    r_x[i], r_y[i],
                    r_xy[i],
                    r2_x[i], r2_y[i],
                    r2_xy[i]);
    }

    return destination;
}

// generate the algorithm explicitly for...

template float* tgvk_l1_deshade_launch_2d(float* f_host,
                      const uint width, const uint height,
                      const float lambda,
                      const uint iteration_count,
                      const uint paint_iteration_interval,
                      DeshadeIterationCallback2D<float> iteration_finished_callback,
                      const uint order,
                      const float* alpha,
                      float** v_x_host, float**v_y_host);

template double* tgvk_l1_deshade_launch_2d(double* f_host,
                      const uint width, const uint height,
                      const double lambda,
                      const uint iteration_count,
                      const uint paint_iteration_interval,
                      DeshadeIterationCallback2D<double> iteration_finished_callback,
                      const uint order,
                      const double* alpha,
                      double** v_x_host, double**v_y_host);
