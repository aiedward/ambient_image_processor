
#include <hip/hip_runtime.h>

typedef const unsigned int DimensionSize;

template<typename Pixel>
Pixel  mean(Pixel* image, DimensionSize voxel_count)
{
    Pixel sum = 0;
    for(int i = 0; i < voxel_count; i++)
        sum+= image[i];
    return sum / voxel_count;
}

template<typename Pixel>
Pixel  standard_deviation(Pixel* image, DimensionSize voxel_count, Pixel mean)
{
    Pixel sum = 0;
    for(int i = 0; i < voxel_count; i++)
    {
        Pixel difference = image[i] - mean;
        sum+= difference * difference;
    }
    return std::sqrt(sum / (voxel_count - 1)); // TODO sqrt
}

template<typename Pixel>
Pixel  normalized_cross_correlation(
    Pixel* image1, Pixel* image2,
    DimensionSize voxel_count)
{
    const auto mean1 = mean(image1, voxel_count);
    const auto std1 = standard_deviation(image1, voxel_count, mean1);
    const auto mean2 = mean(image1, voxel_count);
    const auto std2 = standard_deviation(image2, voxel_count, mean2);

    Pixel normalized_cross_correlation = 0;
    for(int i = 0; i < voxel_count; i++)
    {
        const auto difference1 = image1[i] - mean1;
        const auto difference2 = image2[i] - mean2;
        normalized_cross_correlation+= difference1 * difference2;
    }
    return normalized_cross_correlation / (std1 * std2 * voxel_count);
}

template<typename Pixel>
Pixel  sum_of_absolute_differences(Pixel* image1, Pixel* image2, DimensionSize voxel_count)
{
    Pixel abs_change_sum = 0;
    for(int i = 0; i < voxel_count; i++)
    {
        abs_change_sum+= std::abs(image1[i] - image2[i]);
    }
    return abs_change_sum;
}
