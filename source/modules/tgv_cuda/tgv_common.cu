#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"

#include <functional>

template<typename Pixel>
using IterationCallback = std::function<void(uint iteration_index, uint iteration_count, Pixel* u)>;

#include "cuda_helper.cuh"


template<typename Pixel>
__global__ void clone2(
        Pixel* f, Pixel* u, Pixel* u_bar,
        uint voxel_count) {

    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= voxel_count)
        return;

    u[index] = u_bar[index] = f[index];
}

template<typename Pixel>
__global__ void zeroInit(
        Pixel* p_x, Pixel* p_y, Pixel* p_z,
        Pixel* p_xx, Pixel* p_yy, Pixel* p_zz,
        uint voxel_count, uint depth) {

    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= voxel_count)
        return;

    p_x[index] = p_y[index] =
    p_xx[index] = p_yy[index] = 0;

    if(depth > 1)
        p_z[index] = p_zz[index] = 0;
}

template<typename Pixel>
__global__ void forward_difference_x(
        Pixel* u_bar, Pixel* p_x, const uint width, const uint height, const uint depth) {

    const uint index1 = blockDim.x * blockIdx.x + threadIdx.x;

    if(index1 >= height*depth)
        return;

    const uint y = floorf(index1 / depth);
    const uint z = index1 - y*depth;

    const uint offset = z*width*height + y*width;

    p_x[offset + width - 1] = 0; // neumann boundary condition
    for(uint x = 0; x < width - 1; x++)
    {
        const uint index2 = offset + x;
        p_x[index2] = u_bar[index2 + 1] - u_bar[index2];
    }
}

template<typename Pixel>
__global__ void forward_difference_y(
        Pixel* u_bar, Pixel* p_y, const uint width, const uint height, const uint depth) {

    const uint index1 = blockDim.x * blockIdx.x + threadIdx.x;

    if(index1 >= width*depth)
        return;

    const uint x = floorf(index1 / depth);
    const uint z = index1 - x*depth;

    const uint offset = z*width*height + x;

    p_y[offset + (height - 1) * width] = 0; // neumann boundary condition
    for(uint y = 0; y < height - 1; y++)
    {
        const uint index2 = offset + y * width;
        p_y[index2] = u_bar[index2 + width] - u_bar[index2];
    }
}

template<typename Pixel>
__global__ void forward_difference_z(
        Pixel* u_bar, Pixel* p_z, const uint width, const uint height, const uint depth) {

    const uint index1 = blockDim.x * blockIdx.x + threadIdx.x;

    const uint width_x_height = width * height;
    if(index1 >= width_x_height)
        return;

    const uint y = floorf(index1 / width);
    const uint x = index1 - y*width;

    const uint offset = y*width + x;

    p_z[offset + (depth - 1) * width_x_height] = 0; // neumann boundary condition
    for(uint z = 0; z < depth - 1; z++)
    {
        const uint index2 = offset + z * width_x_height;
        p_z[index2] = u_bar[index2 + width_x_height] - u_bar[index2];
    }
}

template<typename Pixel>
__global__ void backward_difference_x(
        Pixel* u_bar, Pixel* p_x, const uint width, const uint height, const uint depth) {

    const uint index1 = blockDim.x * blockIdx.x + threadIdx.x;

    if(index1 >= height*depth)
        return;

    const uint y = floorf(index1 / depth);
    const uint z = index1 - y*depth;

    const uint offset = z*width*height + y*width;

    p_x[offset] = - u_bar[offset]; // neumann boundary condition of gradient
    for(uint x = 1; x < width; x++)
    {
        const uint index2 = offset + x;
        p_x[index2] = - u_bar[index2] + u_bar[index2 - 1];  // note: the sign
    }
}

template<typename Pixel>
__global__ void backward_difference_y(
        Pixel* u_bar, Pixel* p_y, const uint width, const uint height, const uint depth) {

    const uint index1 = blockDim.x * blockIdx.x + threadIdx.x;

    if(index1 >= width*depth)
        return;

    const uint x = floorf(index1 / depth);
    const uint z = index1 - x*depth;

    const uint offset = z*width*height + x;

    p_y[offset] = - u_bar[offset]; // neumann boundary condition
    for(uint y = 1; y < height; y++)
    {
        const uint index2 = offset + y * width;
        p_y[index2] = - u_bar[index2] + u_bar[index2 - width] ;
    }
}

template<typename Pixel>
__global__ void backward_difference_z(
        Pixel* u_bar, Pixel* p_z, const uint width, const uint height, const uint depth) {

    const uint index1 = blockDim.x * blockIdx.x + threadIdx.x;

    const uint width_x_height = width * height;
    if(index1 >= width_x_height)
        return;

    const uint y = floorf(index1 / width);
    const uint x = index1 - y*width;

    const uint offset = y*width + x;

    p_z[offset] = - u_bar[offset]; // neumann boundary condition
    for(uint z = 1; z < depth; z++)
    {
        const uint index2 = offset + z * width_x_height;
        p_z[index2] = - u_bar[index2] + u_bar[index2 - width_x_height];
    }
}



template<typename Pixel>
__global__ void tgv_kernel_part2(
        Pixel* p_x, Pixel* p_y, Pixel* p_z,
        Pixel* p_xx, Pixel* p_yy, Pixel* p_zz,
        const Pixel sigma, const Pixel alpha1, Pixel* u_previous, Pixel* u,
        const uint width, const uint height, const uint depth) {

    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height * depth)
        return;

    /*
    * Matlab Code:
    p = p + sigma*nabla*u_bar;
    norm_p = sqrt(p(1:N).^2 + p(N+1:2*N).^2);
    p = p./max(1,[norm_p; norm_p]);

    u_old = u;
    */

    p_xx[index] += sigma * p_x[index];
    p_yy[index] += sigma * p_y[index];
    if(depth > 1)
        p_zz[index] += sigma * p_z[index];

    Pixel normalization =
            p_xx[index] * p_xx[index] +
            p_yy[index] * p_yy[index];
    if(depth > 1)
        normalization += p_zz[index] * p_zz[index];

    normalization = fmax(alpha1, sqrt(normalization));

    p_xx[index] /= normalization;
    p_yy[index] /= normalization;
    if(depth > 1)
        p_zz[index] /= normalization;

    u_previous[index] = u[index];
}


template<typename Pixel>
void tgv_launch_part1(
          uint width, uint height, uint depth,
          uint &voxel_count,
          dim3 &block_dimension,
          dim3 &grid_dimension,
          dim3 &grid_dimension_x,
          dim3 &grid_dimension_y,
          dim3 &grid_dimension_z)
{
    int cuda_device_count;
    cudaCheckError( hipGetDeviceCount(&cuda_device_count) );

    printf("found %d cuda devices.\n", cuda_device_count);

    voxel_count = width*height*depth;

    block_dimension = dim3(CUDA_BLOCK_DIMENSON);
    grid_dimension = dim3((voxel_count + block_dimension.x - 1) / block_dimension.x);

    grid_dimension_x = dim3((depth*height + block_dimension.x - 1) / block_dimension.x);
    grid_dimension_y = dim3((depth*width + block_dimension.x - 1) / block_dimension.x);
    grid_dimension_z = dim3((width*height + block_dimension.x - 1) / block_dimension.x);

    printf("block dimensions: x:%d, y:%d \n", block_dimension.x);
    printf("grid dimensions: x:%d, y:%d \n", grid_dimension.x);
}

template<typename Pixel>
void tgv_launch_part2(Pixel* f_host,
          uint voxel_count, uint depth,
          Pixel** f, Pixel** u,
          Pixel** u_previous, Pixel**u_bar,
          Pixel**p_x, Pixel**p_y, Pixel**p_z,
          Pixel**p_xx, Pixel**p_yy, Pixel**p_zz) {

    printf("voxel_count: %d \n", voxel_count);

    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMallocManaged(f, size) )
    cudaCheckError( hipMemcpy(*f, f_host, size, hipMemcpyHostToDevice) )

    cudaCheckError( hipMallocManaged(u, size) )
    cudaCheckError( hipMallocManaged(u_previous, size) )
    cudaCheckError( hipMallocManaged(u_bar, size) )
    cudaCheckError( hipMallocManaged(p_x, size) )
    cudaCheckError( hipMallocManaged(p_y, size) )
    cudaCheckError( hipMallocManaged(p_xx, size) )
    cudaCheckError( hipMallocManaged(p_yy, size) )
    if(depth > 1) {
        cudaCheckError( hipMallocManaged(p_z, size) )
        cudaCheckError( hipMallocManaged(p_zz, size) )
    }
}

template<typename Pixel>
void tgv_launch_forward_differences(Pixel* u_bar,
        Pixel* p_x, Pixel* p_y, Pixel* p_z,
        uint width, uint height, uint depth,
        dim3 block_dimension,
        dim3 grid_dimension_x,
        dim3 grid_dimension_y,
        dim3 grid_dimension_z)
{
    forward_difference_x<<<grid_dimension_x, block_dimension>>>(
          u_bar, p_x, width, height, depth);
    forward_difference_y<<<grid_dimension_y, block_dimension>>>(
          u_bar, p_y, width, height, depth);
    if(depth > 1)
        forward_difference_z<<<grid_dimension_z, block_dimension>>>(
             u_bar, p_z, width, height, depth);
    cudaCheckError( hipDeviceSynchronize() );
}

template<typename Pixel>
void tgv_launch_backward_differences(
        Pixel* p_x, Pixel* p_y, Pixel* p_z,
        Pixel* p_xx, Pixel* p_yy, Pixel* p_zz,
        uint width, uint height, uint depth,
        dim3 block_dimension,
        dim3 grid_dimension_x,
        dim3 grid_dimension_y,
        dim3 grid_dimension_z)
{
    backward_difference_x<<<grid_dimension_x, block_dimension>>>(
                                                                   p_xx, p_x, width, height, depth);
    backward_difference_y<<<grid_dimension_y, block_dimension>>>(
                                                                   p_yy, p_y, width, height, depth);
    if(depth > 1)
        backward_difference_z<<<grid_dimension_z, block_dimension>>>(
                                                                   p_zz, p_z, width, height, depth);
    cudaCheckError( hipDeviceSynchronize() );
}


template<typename Pixel>
void tgv_launch_part3(
            Pixel* host_f,
            uint voxel_count, uint depth,
            Pixel* u_previous, Pixel* u_bar,
            Pixel* p_x, Pixel* p_y, Pixel* p_z,
            Pixel* p_xx, Pixel* p_yy, Pixel* p_zz,
            Pixel* f, Pixel* u)
{
    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMemcpy(host_f, u, size, hipMemcpyDeviceToHost) );
    cudaCheckError( hipDeviceSynchronize() );

    hipFree(u_previous);
    hipFree(u_bar);
    hipFree(p_x);
    hipFree(p_y);
    hipFree(p_xx);
    hipFree(p_yy);
    if(depth > 1) {
        hipFree(p_z);
        hipFree(p_zz);
    }
    hipFree(f);
    hipFree(u);
}
