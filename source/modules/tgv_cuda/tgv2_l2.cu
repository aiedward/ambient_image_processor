#include "hip/hip_runtime.h"
/*
    Ambient Image Processor - A tool to perform several imaging tasks
    
    Copyright (C) 2016 Josef Koller

    https://github.com/josefkoller/ambient_image_processor    
    
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "tgv_common.cu"
#include "tgv2_common.cu"

template<typename Pixel>
__global__ void tgv_kernel_part4_tgv2_l2(
        Pixel* p_x, Pixel* p_y, Pixel* p_z,
        const Pixel tau, Pixel* u, Pixel* f,
        const Pixel lambda,
        Pixel* u_previous, const Pixel theta, Pixel* u_bar,
        const uint width, const uint height, const uint depth)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height * depth)
        return;

    if(depth > 1)
        u[index] -= tau * (p_x[index] + p_y[index] + p_z[index]);
    else
        u[index] -= tau * (p_x[index] + p_y[index]);

    u[index] = (u[index] + tau*lambda*f[index]) / (1 + tau*lambda);

    u_bar[index] = u[index] + theta*(u[index] - u_previous[index]);

    /*
     *  Matlab Code:
          u = u - tau * nabla_t * p;
          u = (u + tau * lambda .* f)/(1 + tau * lambda);

          % overrelaxation
          u_bar = u + theta*(u - u_old);
    */
}


template<typename Pixel>
Pixel* tgv2_l2_launch(Pixel* f_host,
                  uint width, uint height, uint depth,
                  Pixel lambda,
                  uint iteration_count,
                  uint paint_iteration_interval,
                  IterationCallback<Pixel> iteration_finished_callback,
                  Pixel alpha0,
                  Pixel alpha1)
{
    /* MATLAB Code
     *
        nabla_second = [nabla_x, zeros_sparse, zeros_sparse, nabla_y/2, nabla_z/2, zeros_sparse;...
            zeros_sparse, nabla_y, zeros_sparse, nabla_x/2, zeros_sparse, nabla_z/2;...
            zeros_sparse, zeros_sparse, nabla_z, zeros_sparse, nabla_x/2, nabla_y/2]';

        nabla_second_t = [nabla_x, zeros_sparse, zeros_sparse, nabla_y, nabla_z, zeros_sparse;...
            zeros_sparse, nabla_y, zeros_sparse, nabla_x, zeros_sparse, nabla_z;...
            zeros_sparse, zeros_sparse, nabla_z, zeros_sparse, nabla_x, nabla_y];

        % fixed parameters
        L = sqrt(20);  % Lipschitz constant ? this is an approx. to normest()
        tau    = 1/L;
        sigma  = 1/L;

        theta  = 1;

        % initializations
        p = zeros(3*N, 1);
        v = zeros(3*N, 1);
        q = zeros(6*N, 1);

        u = f;
        u_bar = u;  % overrelaxation u
        v_bar = v;
        nabla_t = nabla';
        %nabla_second_t = nabla_second';

        for currIter = 1:maxIter
            u_old = u;
            v_old = v;

            % dual update p
            p = p + sigma*(nabla*u_bar - v_bar);
            norm_p  = sqrt(p(1:N).^2 + p(N+1:2*N).^2 +  p(2*N+1:3*N).^2);
            p = p./max(1,[norm_p; norm_p; norm_p]/alpha1);

            % dual update q
            q = q + sigma*nabla_second*v_bar;
            norm_q = sqrt(q(1:N).^2 + q(N+1:2*N).^2 + q(2*N+1:3*N).^2 + ... % main diagonal
                2*q(3*N+1:4*N).^2 + 2*q(4*N+1:5*N).^2 + 2*q(5*N+1:6*N).^2); % off diagonal
            q = q./max(1, repmat(norm_q, 6, 1)/alpha0);

            % primal update u
            u = u - tau * nabla_t * p;

            % projection of u
            index1 = (u - f) > tau*lambda;
            index2 = (u - f) < -tau*lambda;
            index3 = abs(u - f) <= tau*lambda;

            u(index1) = u(index1) - tau*lambda;
            u(index2) = u(index2) + tau*lambda;
            u(index3) = f(index3);

            % overrelaxation u
            u_bar = u + theta*(u - u_old);

            % primal update v
            v = v - tau * (nabla_second_t * q - p);
            v_bar = v + theta*(v - v_old);
            */
    uint voxel_count;
    dim3 block_dimension;
    dim3 grid_dimension;
    dim3 grid_dimension_x;
    dim3 grid_dimension_y;
    dim3 grid_dimension_z;

    tgv_launch_part1<Pixel>(
                        width, height, depth,
                        voxel_count,
                        block_dimension,
                        grid_dimension,
                        grid_dimension_x,
                        grid_dimension_y,
                        grid_dimension_z);

    Pixel* f, *u;
    Pixel* u_previous, *u_bar, *p_x, *p_y, *p_z, *p_xx, *p_yy, *p_zz;

    tgv_launch_part2<Pixel>(f_host,
                voxel_count,
                &f, &u,
                &u_previous, &u_bar, &p_x, &p_y, &p_z, &p_xx, &p_yy, &p_zz);

    Pixel *v_x, *v_y, *v_z;
    Pixel *v_bar_x, *v_bar_y, *v_bar_z;
    Pixel *v_previous_x, *v_previous_y, *v_previous_z;
    Pixel *q_x, *q_y, *q_z, *q_xy, *q_xz, *q_yz;
    Pixel *q_x2, *q_y2, *q_z2, *q_xy2, *q_xz2, *q_yz2;
    Pixel * q_temp;

    tgv_launch_part22<Pixel>(
                voxel_count,
                &v_previous_x, &v_previous_y, &v_previous_z,
                &v_bar_x, &v_bar_y, &v_bar_z,
                &v_x, &v_y, &v_z,
                &q_x, &q_y, &q_z,
                &q_xy, &q_xz, &q_yz,
                &q_x2, &q_y2, &q_z2,
                &q_xy2, &q_xz2, &q_yz2, &q_temp);

    // algorithm variables..
    const Pixel tau = 1.0 / std::sqrt(20.0);
    const Pixel sigma = tau;
    const Pixel theta = 1;

    // algorithm begin
    zeroInit<<<grid_dimension, block_dimension>>>(
                                                    p_x, p_y, p_z,
                                                    p_xx, p_yy, p_zz,
                                                    voxel_count);
    zeroInit<<<grid_dimension, block_dimension>>>(
                                                    v_x, v_y, v_z,
                                                    v_bar_x, v_bar_y, v_bar_z,
                                                    voxel_count);
    zeroInit2<<<grid_dimension, block_dimension>>>(
                                                    q_x, q_y, q_z,
                                                    q_xy, q_xz, q_yz,
                                                    voxel_count);
    clone2<<<grid_dimension, block_dimension>>>(
                                                  f, u, u_bar, voxel_count);
    cudaCheckError( hipDeviceSynchronize() );

    for(uint iteration_index = 0; iteration_index < iteration_count; iteration_index++)
    {
        tgv_launch_forward_differences<Pixel>(u_bar,
                p_x,p_y,p_z,
                width, height, depth,
                block_dimension,
                grid_dimension_x,
                grid_dimension_y,
                grid_dimension_z);

        tgv_kernel_part22<<<grid_dimension, block_dimension>>>( v_bar_x, v_bar_y, v_bar_z,
                                                                p_x, p_y, p_z,
                                                                p_xx, p_yy, p_zz,
                                                                sigma, alpha1, u_previous, u,
                                                                width, height, depth);
        cudaCheckError( hipDeviceSynchronize() );

        tgv_launch_backward_differences<Pixel>(
                p_x, p_y, p_z,
                p_xx, p_yy, p_zz,
                width, height, depth,
                block_dimension,
                grid_dimension_x,
                grid_dimension_y,
                grid_dimension_z);
        cudaCheckError( hipDeviceSynchronize() );

        tgv_kernel_part4_tgv2_l2<<<grid_dimension, block_dimension>>>(
                                                                p_x, p_y, p_z,
                                                                tau, u, f,
                                                                lambda,
                                                                u_previous, theta, u_bar,
                                                                width, height, depth);
        cudaCheckError( hipDeviceSynchronize() );

        /*
        % dual update q
        q = q + sigma*nabla_second*v_bar;
        norm_q = sqrt(q(1:N).^2 + q(N+1:2*N).^2 + q(2*N+1:3*N).^2 + ... % main diagonal
            2*q(3*N+1:4*N).^2 + 2*q(4*N+1:5*N).^2 + 2*q(5*N+1:6*N).^2); % off diagonal
        q = q./max(1, repmat(norm_q, 6, 1)/alpha0);

            v_old = v;
        */
        tgv_launch_gradient2<Pixel>(
                v_bar_x, v_bar_y, v_bar_z,
                q_x2,q_y2,q_z2,
                q_xy2,q_xz2,q_yz2, q_temp,
                width, height, depth,
                block_dimension,
                grid_dimension,
                grid_dimension_x,
                grid_dimension_y,
                grid_dimension_z);

        tgv_kernel_part5<<<grid_dimension, block_dimension>>>(
                v_x, v_y, v_z,
                v_previous_x, v_previous_y, v_previous_z,
                q_x2,q_y2,q_z2,
                q_xy2,q_xz2,q_yz2,
                q_x, q_y, q_z,
                q_xy, q_xz, q_yz,
                sigma, alpha0,
                width, height, depth);
        cudaCheckError( hipDeviceSynchronize() );

        tgv_launch_divergence2<Pixel>(
                q_x, q_y, q_z,
                q_xy, q_xz, q_yz,
                q_x2, q_y2, q_z2,
                q_temp,
                width, height, depth,
                block_dimension,
                grid_dimension,
                grid_dimension_x,
                grid_dimension_y,
                grid_dimension_z);
        cudaCheckError( hipDeviceSynchronize() );


        /*
            % primal update v
            v = v - tau * (nabla_second_t * q - p);
            v_bar = v + theta*(v - v_old);
            */

        tgv_kernel_part6<<<grid_dimension, block_dimension>>>(
                v_x, v_y, v_z,
                q_x2, q_y2, q_z2,
                p_xx, p_yy, p_zz,
                v_previous_x, v_previous_y, v_previous_z,
                v_bar_x, v_bar_y, v_bar_z,
                tau, theta,
                width, height, depth);
        cudaCheckError( hipDeviceSynchronize() );


        bool stop = tgv2_iteration_callback(
                    iteration_index, iteration_count, paint_iteration_interval,
                    u,
                    iteration_finished_callback, voxel_count);
        if(stop)
            break;
    }

    Pixel* destination = new Pixel[voxel_count];
    tgv_launch_part3<Pixel>(
                destination,
                voxel_count, depth,
                u_previous, u_bar,
                p_x, p_y, p_z,
                p_xx, p_yy, p_zz,
                f, u);
    tgv_launch_part32<Pixel>( depth,
              v_bar_x, v_bar_y, v_bar_z,
              v_previous_x, v_previous_y, v_previous_z,
                v_x, v_y, v_z,
                q_x, q_y, q_z,
                q_xy, q_xz, q_yz,
              q_x2, q_y2, q_z2,
              q_xy2, q_xz2, q_yz2, q_temp);

    return destination;
}

// generate the algorithm explicitly for...

template float* tgv2_l2_launch(float* f_host,
uint width, uint height, uint depth,
float lambda,
uint iteration_count,
uint paint_iteration_interval,
IterationCallback<float> iteration_finished_callback,
float alpha0,
float alpha1);

template double* tgv2_l2_launch(double* f_host,
uint width, uint height, uint depth,
double lambda,
uint iteration_count,
uint paint_iteration_interval,
IterationCallback<double> iteration_finished_callback,
double alpha0,
double alpha1);
