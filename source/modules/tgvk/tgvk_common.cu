#include "hip/hip_runtime.h"
/*
    Ambient Image Processor - A tool to perform several imaging tasks
    
    Copyright (C) 2016 Josef Koller

    https://github.com/josefkoller/ambient_image_processor    
    
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef TGV_K_COMMON
#define TGV_K_COMMON

#include "tgv3_common.cu"

template<typename Pixel>
__global__ void tgvk_kernel_part5(
        Pixel* r2_x, Pixel* r2_y, Pixel* r2_z,
        Pixel* r2_xy, Pixel* r2_xz, Pixel* r2_yz,

        Pixel* r_x, Pixel* r_y, Pixel* r_z,
        Pixel* r_xy, Pixel* r_xz, Pixel* r_yz,

        Pixel* w_x, Pixel* w_y, Pixel* w_z,
        Pixel* w_xy, Pixel* w_xz, Pixel* w_yz,

        Pixel* w_previous_x, Pixel* w_previous_y, Pixel* w_previous_z,
        Pixel* w_previous_xy, Pixel* w_previous_xz, Pixel* w_previous_yz,

        Pixel* w_prime_x, Pixel* w_prime_y, Pixel* w_prime_z,
        Pixel* w_prime_xy, Pixel* w_prime_xz, Pixel* w_prime_yz,

        const Pixel sigma, const Pixel alpha2,
        const uint width, const uint height, const uint depth) {

    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height * depth)
        return;

    r_x[index] = fmaf(sigma, r2_x[index] - w_prime_x[index], r_x[index]);
    r_y[index] = fmaf(sigma, r2_y[index] - w_prime_y[index], r_y[index]);
    r_xy[index] = fmaf(sigma, r2_xy[index] - w_prime_xy[index], r_xy[index]);

    r_z[index] = fmaf(sigma, r2_z[index] - w_prime_z[index], r_z[index]);
    r_xz[index] = fmaf(sigma, r2_xz[index] - w_prime_xz[index], r_xz[index]);
    r_yz[index] = fmaf(sigma, r2_yz[index] - w_prime_yz[index], r_yz[index]);

    Pixel normalization =
            r_x[index] * r_x[index] +
            r_y[index] * r_y[index] +
            2 * r_xy[index] * r_xy[index] +
            r_z[index] * r_z[index] +
            2 * r_xz[index] * r_xz[index] +
            2 * r_yz[index] * r_yz[index];

    normalization = fmaxf(1, sqrtf(normalization) / alpha2);

    r_x[index] /= normalization;
    r_y[index] /= normalization;
    r_xy[index] /= normalization;

    r_z[index] /= normalization;
    r_xz[index] /= normalization;
    r_yz[index] /= normalization;

    w_previous_z[index] = w_z[index];
    w_previous_xz[index] = w_xz[index];
    w_previous_yz[index] = w_yz[index];

    w_previous_x[index] = w_x[index];
    w_previous_y[index] = w_y[index];
    w_previous_xy[index] = w_xy[index];
}


template<typename Pixel>
void tgv_launch_gradient3_backward(
        Pixel* w_bar_x, Pixel* w_bar_y, Pixel* w_bar_z,
        Pixel* w_bar_xy, Pixel* w_bar_xz, Pixel* w_bar_yz,

        Pixel* r_x, Pixel* r_y, Pixel* r_z,
        Pixel* r_xy, Pixel* r_xz, Pixel* r_yz,  Pixel* q_temp,
        uint width, uint height, uint depth,
        dim3 block_dimension,
        dim3 grid_dimension,
        dim3 grid_dimension_x,
        dim3 grid_dimension_y,
        dim3 grid_dimension_z)
{
    backward_difference_x<<<grid_dimension_x, block_dimension>>>(
          w_bar_x, r_x, width, height, depth);

    backward_difference_y<<<grid_dimension_y, block_dimension>>>(
          w_bar_y, r_y, width, height, depth);

    backward_difference_x<<<grid_dimension_x, block_dimension>>>(
          w_bar_xy, r_xy, width, height, depth);
    backward_difference_y<<<grid_dimension_y, block_dimension>>>(
          w_bar_xy, q_temp, width, height, depth);
    addAndHalf<<<grid_dimension, block_dimension>>>(
            r_xy, q_temp, r_xy,
            width, height, depth);

    backward_difference_z<<<grid_dimension_z, block_dimension>>>(
          w_bar_z, r_z, width, height, depth);

    backward_difference_x<<<grid_dimension_x, block_dimension>>>(
          w_bar_xz, r_xz, width, height, depth);
    backward_difference_z<<<grid_dimension_z, block_dimension>>>(
          w_bar_xz, q_temp, width, height, depth);
    addAndHalf<<<grid_dimension, block_dimension>>>(
            r_xz, q_temp, r_xz,
            width, height, depth);

    backward_difference_y<<<grid_dimension_y, block_dimension>>>(
          w_bar_yz, r_yz, width, height, depth);
    backward_difference_z<<<grid_dimension_z, block_dimension>>>(
          w_bar_yz, q_temp, width, height, depth);
    addAndHalf<<<grid_dimension, block_dimension>>>(
            r_yz, q_temp, r_yz,
            width, height, depth);
    cudaCheckError( hipDeviceSynchronize() );
}


template<typename Pixel>
void tgv_launch_divergence3_forward(
        Pixel* r_x, Pixel* r_y, Pixel* r_z,
        Pixel* r_xy, Pixel* r_xz, Pixel* r_yz,

        Pixel* r2_x, Pixel* r2_y, Pixel* r2_z,
        Pixel* r2_xy, Pixel* r2_xz, Pixel* r2_yz,

        Pixel* r_temp,

        uint width, uint height, uint depth,
        dim3 block_dimension,
        dim3 grid_dimension,
        dim3 grid_dimension_x,
        dim3 grid_dimension_y,
        dim3 grid_dimension_z)
{
    forward_difference_x<<<grid_dimension_x, block_dimension>>>(
      r_x, r2_x, width, height, depth);

    forward_difference_y<<<grid_dimension_y, block_dimension>>>(
      r_y, r2_y, width, height, depth);

    forward_difference_x<<<grid_dimension_x, block_dimension>>>(
      r_xy, r2_xy, width, height, depth);
    forward_difference_y<<<grid_dimension_y, block_dimension>>>(
      r_xy, r_temp, width, height, depth);
    add<<<grid_dimension, block_dimension>>>(r2_xy, r_temp, r2_xy, width, height, depth);

    forward_difference_z<<<grid_dimension_z, block_dimension>>>(
      r_z, r2_z, width, height, depth);

    forward_difference_x<<<grid_dimension_x, block_dimension>>>(
      r_xz, r2_xz, width, height, depth);
    forward_difference_z<<<grid_dimension_z, block_dimension>>>(
      r_xz, r_temp, width, height, depth);
    add<<<grid_dimension, block_dimension>>>(r2_xz, r_temp, r2_xz, width, height, depth);

    forward_difference_y<<<grid_dimension_y, block_dimension>>>(
      r_yz, r2_yz, width, height, depth);
    forward_difference_z<<<grid_dimension_z, block_dimension>>>(
      r_yz, r_temp, width, height, depth);
    add<<<grid_dimension, block_dimension>>>(r2_yz, r_temp, r2_yz, width, height, depth);
    cudaCheckError( hipDeviceSynchronize() );
}

#endif // TGV_K_COMMON
