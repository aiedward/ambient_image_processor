

#include "cuda_helper.cuh"

template<typename Pixel>
void unary_operation_part1(Pixel* image_host,
                  uint width, uint height, uint depth,
                  Pixel** image,
                  dim3& block_dimension,
                  dim3& grid_dimension)
{
    int cuda_device_count;
    cudaCheckError( hipGetDeviceCount(&cuda_device_count) );

//    printf("found %d cuda devices.\n", cuda_device_count);

    uint voxel_count = width*height*depth;
    block_dimension = dim3(CUDA_BLOCK_DIMENSON);
    grid_dimension = dim3((voxel_count + block_dimension.x - 1) / block_dimension.x);

    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMalloc(image, size) )
    cudaCheckError( hipMemcpy(*image, image_host, size, hipMemcpyHostToDevice) )
    cudaCheckError( hipDeviceSynchronize() );
}


template<typename Pixel>
Pixel* unary_operation_part2(Pixel* image,
                  uint width, uint height, uint depth)
{
    uint voxel_count = width*height*depth;
    Pixel* result = new Pixel[voxel_count];

    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMemcpy(result, image, size, hipMemcpyDeviceToHost) );
    cudaCheckError( hipDeviceSynchronize() );

    cudaCheckError( hipFree(image) );

    return result;
}
