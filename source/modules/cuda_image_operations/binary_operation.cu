#include "cuda_helper.cuh"

template<typename Pixel>
void binary_operation_part1(Pixel* image1_host, Pixel* image2_host,
                  uint width, uint height, uint depth,
                  Pixel** image1, Pixel** image2,
                  dim3& block_dimension,
                  dim3& grid_dimension)
{
    int cuda_device_count;
    cudaCheckError( hipGetDeviceCount(&cuda_device_count) );

//    printf("found %d cuda devices.\n", cuda_device_count);

    uint voxel_count = width*height*depth;
    block_dimension = dim3(CUDA_BLOCK_DIMENSON);
    grid_dimension = dim3((voxel_count + block_dimension.x - 1) / block_dimension.x);

    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMalloc(image1, size) )
    cudaCheckError( hipMemcpy(*image1, image1_host, size, hipMemcpyHostToDevice) )
    cudaCheckError( hipMalloc(image2, size) )
    cudaCheckError( hipMemcpy(*image2, image2_host, size, hipMemcpyHostToDevice) )

    cudaCheckError( hipDeviceSynchronize() );
}


template<typename Pixel>
Pixel* binary_operation_part2(Pixel* image1, Pixel* image2,
                  uint width, uint height, uint depth)
{
    uint voxel_count = width*height*depth;
    Pixel* result = new Pixel[voxel_count];

    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMemcpy(result, image1, size, hipMemcpyDeviceToHost) );
    cudaCheckError( hipDeviceSynchronize() );

    cudaCheckError( hipFree(image1) );
    cudaCheckError( hipFree(image2) );

    return result;
}
