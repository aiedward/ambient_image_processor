#include "hip/hip_runtime.h"
/*
    Ambient Image Processor - A tool to perform several imaging tasks
    
    Copyright (C) 2016 Josef Koller

    https://github.com/josefkoller/ambient_image_processor    
    
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "cuda_helper.cuh"

template<typename Pixel>
__global__  void add_constant_kernel(Pixel* image,
                              uint width, uint height, uint depth,
                              Pixel constant, Pixel* result)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height * depth)
        return;

    result[index] = image[index] + constant;
}

template<typename Pixel>
Pixel* add_constant_kernel_launch(Pixel* image_host,
                              uint width, uint height, uint depth,
                              Pixel constant_host)
{
    int cuda_device_count;
    cudaCheckError( hipGetDeviceCount(&cuda_device_count) );

//    printf("found %d cuda devices.\n", cuda_device_count);

    uint voxel_count = width*height*depth;
    dim3 block_dimension(CUDA_BLOCK_DIMENSON);
    dim3 grid_dimension((voxel_count + block_dimension.x - 1) / block_dimension.x);

    Pixel* image, *result;

    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMalloc(&image, size) )
    cudaCheckError( hipMemcpy(image, image_host, size, hipMemcpyHostToDevice) )

    cudaCheckError( hipMalloc(&result, size) )
    cudaCheckError( hipDeviceSynchronize() );


    add_constant_kernel<<<grid_dimension, block_dimension>>>(
      image, width, height, depth,
      constant_host, result);
    cudaCheckError( hipDeviceSynchronize() );

    Pixel* result_host = new Pixel[voxel_count];
    cudaCheckError( hipMemcpy(result_host, result, size, hipMemcpyDeviceToHost) );
    cudaCheckError( hipDeviceSynchronize() );

    hipFree(image);
    hipFree(result);

    return result_host;
}

template float* add_constant_kernel_launch(float* image_host,
                              uint width, uint height, uint depth,
                              float constant_host);
template double* add_constant_kernel_launch(double* image_host,
                              uint width, uint height, uint depth,
                              double constant_host);
