#include "hip/hip_runtime.h"
#include "tgv_common_2d.cu"

template<typename Pixel>
__global__ void add_kernel_2d(
        Pixel* q_xx, Pixel* q_yx,
        uint width, uint height) {
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width*height)
        return;

    q_xx[index] = q_xx[index] + q_yx[index];
}

template<typename Pixel>
void launch_divergence_2d(
        Pixel* dx, Pixel* dy,
        Pixel* dxdx, Pixel* dydy,

        const uint width, const uint height,

        dim3 block_dimension,
        dim3 grid_dimension,
        dim3 grid_dimension_x,
        dim3 grid_dimension_y)
{
    tgv_launch_backward_differences_2d<Pixel>(
            dxdx, dydy,
            dx, dy,
            width, height,
            block_dimension,
            grid_dimension_x,
            grid_dimension_y);

    add_kernel_2d<<<grid_dimension, block_dimension>>>(
         dxdx, dydy, width, height);
    cudaCheckError( hipDeviceSynchronize() );
}

template<typename Pixel>
Pixel* divergence_2d_kernel_launch(
        Pixel* dx, Pixel* dy,
        const uint width, const uint height, bool is_host_data=false)
{
    uint voxel_count;
    dim3 block_dimension;
    dim3 grid_dimension;
    dim3 grid_dimension_x;
    dim3 grid_dimension_y;

    tgv_launch_part1_2d<Pixel>(
                        width, height,
                        voxel_count,
                        block_dimension,
                        grid_dimension,
                        grid_dimension_x,
                        grid_dimension_y);

    if(is_host_data)
    {
        uint voxel_count = width*height;
        size_t size = sizeof(Pixel) * voxel_count;

        Pixel *dx2, *dy2;
        cudaCheckError( hipMalloc(&dx2, size) )
        cudaCheckError( hipMalloc(&dy2, size) )
        cudaCheckError( hipDeviceSynchronize() );

        cudaCheckError( hipMemcpy(dx2, dx, size, hipMemcpyHostToDevice) )
        dx = dx2;
        cudaCheckError( hipMemcpy(dy2, dy, size, hipMemcpyHostToDevice) )
        dy = dy2;
        cudaCheckError( hipDeviceSynchronize() );
    }

    Pixel *dxdx, *dydy;
    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMalloc(&dxdx, size) )
    cudaCheckError( hipMalloc(&dydy, size) )
    cudaCheckError( hipDeviceSynchronize() );

    launch_divergence_2d(dx, dy,
                      dxdx, dydy,
                      width, height,
                      block_dimension,
                      grid_dimension,
                      grid_dimension_x,
                      grid_dimension_y);

    Pixel* result = new Pixel[voxel_count];
    cudaCheckError( hipMemcpy(result, dxdx, size, hipMemcpyDeviceToHost) )
    cudaCheckError( hipDeviceSynchronize() );

    hipFree(dxdx);
    hipFree(dydy);
    cudaCheckError( hipDeviceSynchronize() );

    if(is_host_data)
    {
        hipFree(dx);
        hipFree(dy);
    }

    return result;
}

template float* divergence_2d_kernel_launch(
float* dx, float* dy,
const uint width, const uint height, bool is_host_data);

template double* divergence_2d_kernel_launch(
double* dx, double* dy,
const uint width, const uint height, bool is_host_data);
