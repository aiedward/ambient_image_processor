#include "hip/hip_runtime.h"
/*
    Ambient Image Processor - A tool to perform several imaging tasks
    
    Copyright (C) 2016 Josef Koller

    https://github.com/josefkoller/ambient_image_processor    
    
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "tgv_common_2d.cu"

template<typename Pixel>
__global__ void add_kernel_2d(
        Pixel* q_xx, Pixel* q_yx,
        uint width, uint height) {
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width*height)
        return;

    q_xx[index] = q_xx[index] + q_yx[index];
}

template<typename Pixel>
void launch_divergence_2d(
        Pixel* dx, Pixel* dy,
        Pixel* dxdx, Pixel* dydy,

        const uint width, const uint height,

        dim3 block_dimension,
        dim3 grid_dimension,
        dim3 grid_dimension_x,
        dim3 grid_dimension_y)
{
    tgv_launch_backward_differences_2d<Pixel>(
            dxdx, dydy,
            dx, dy,
            width, height,
            block_dimension,
            grid_dimension_x,
            grid_dimension_y);

    add_kernel_2d<<<grid_dimension, block_dimension>>>(
         dxdx, dydy, width, height);
    cudaCheckError( hipDeviceSynchronize() );
}

template<typename Pixel>
Pixel* divergence_2d_kernel_launch(
        Pixel* dx, Pixel* dy,
        const uint width, const uint height, bool is_host_data=false)
{
    uint voxel_count;
    dim3 block_dimension;
    dim3 grid_dimension;
    dim3 grid_dimension_x;
    dim3 grid_dimension_y;

    tgv_launch_part1_2d<Pixel>(
                        width, height,
                        voxel_count,
                        block_dimension,
                        grid_dimension,
                        grid_dimension_x,
                        grid_dimension_y);

    if(is_host_data)
    {
        uint voxel_count = width*height;
        size_t size = sizeof(Pixel) * voxel_count;

        Pixel *dx2, *dy2;
        cudaCheckError( hipMalloc(&dx2, size) )
        cudaCheckError( hipMalloc(&dy2, size) )
        cudaCheckError( hipDeviceSynchronize() );

        cudaCheckError( hipMemcpy(dx2, dx, size, hipMemcpyHostToDevice) )
        dx = dx2;
        cudaCheckError( hipMemcpy(dy2, dy, size, hipMemcpyHostToDevice) )
        dy = dy2;
        cudaCheckError( hipDeviceSynchronize() );
    }

    Pixel *dxdx, *dydy;
    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMalloc(&dxdx, size) )
    cudaCheckError( hipMalloc(&dydy, size) )
    cudaCheckError( hipDeviceSynchronize() );

    launch_divergence_2d(dx, dy,
                      dxdx, dydy,
                      width, height,
                      block_dimension,
                      grid_dimension,
                      grid_dimension_x,
                      grid_dimension_y);

    Pixel* result = new Pixel[voxel_count];
    cudaCheckError( hipMemcpy(result, dxdx, size, hipMemcpyDeviceToHost) )
    cudaCheckError( hipDeviceSynchronize() );

    hipFree(dxdx);
    hipFree(dydy);
    cudaCheckError( hipDeviceSynchronize() );

    if(is_host_data)
    {
        hipFree(dx);
        hipFree(dy);
    }

    return result;
}

template float* divergence_2d_kernel_launch(
float* dx, float* dy,
const uint width, const uint height, bool is_host_data);

template double* divergence_2d_kernel_launch(
double* dx, double* dy,
const uint width, const uint height, bool is_host_data);
