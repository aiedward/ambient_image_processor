#include "tgv_common.cu"
#include "add.cu"

template<typename Pixel>
void launch_divergence(
        Pixel* dx, Pixel* dy, Pixel* dz,
        Pixel* dxdx, Pixel* dydy, Pixel* dzdz,

        const uint width, const uint height, const uint depth,

        dim3 block_dimension,
        dim3 grid_dimension,
        dim3 grid_dimension_x,
        dim3 grid_dimension_y,
        dim3 grid_dimension_z)
{
    tgv_launch_backward_differences<Pixel>(
            dxdx, dydy, dzdz,
            dx, dy, dz,
            width, height, depth,
            block_dimension,
            grid_dimension_x,
            grid_dimension_y,
            grid_dimension_z);

    add_kernel<<<grid_dimension, block_dimension>>>(
         dxdx, dydy, width, height, depth);
    cudaCheckError( hipDeviceSynchronize() );
    if(depth > 1)
    {
        add_kernel<<<grid_dimension, block_dimension>>>(
             dxdx, dzdz, width, height, depth);
        cudaCheckError( hipDeviceSynchronize() );
    }
}

template<typename Pixel>
Pixel* divergence_kernel_launch(
        Pixel* dx, Pixel* dy, Pixel* dz,
        const uint width, const uint height, const uint depth, bool is_host_data=false)
{
    uint voxel_count;
    dim3 block_dimension;
    dim3 grid_dimension;
    dim3 grid_dimension_x;
    dim3 grid_dimension_y;
    dim3 grid_dimension_z;

    tgv_launch_part1<Pixel>(
                        width, height, depth,
                        voxel_count,
                        block_dimension,
                        grid_dimension,
                        grid_dimension_x,
                        grid_dimension_y,
                        grid_dimension_z);

    if(is_host_data)
    {
        uint voxel_count = width*height*depth;
        size_t size = sizeof(Pixel) * voxel_count;

        Pixel *dx2, *dy2, *dz2;
        cudaCheckError( hipMallocManaged(&dx2, size) )
        cudaCheckError( hipMallocManaged(&dy2, size) )
        if(depth > 1)
          cudaCheckError( hipMallocManaged(&dz2, size) )
        cudaCheckError( hipDeviceSynchronize() );

        cudaCheckError( hipMemcpy(dx2, dx, size, hipMemcpyHostToDevice) )
        dx = dx2;
        cudaCheckError( hipMemcpy(dy2, dy, size, hipMemcpyHostToDevice) )
        dy = dy2;
        if(depth > 1)
        {
            cudaCheckError( hipMemcpy(dz2, dz, size, hipMemcpyHostToDevice) )
            dz = dz2;
        }
        cudaCheckError( hipDeviceSynchronize() );
    }

    Pixel *dxdx, *dydy, *dzdz;
    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMallocManaged(&dxdx, size) )
    cudaCheckError( hipMallocManaged(&dydy, size) )
    if(depth > 1)
      cudaCheckError( hipMallocManaged(&dzdz, size) )
    cudaCheckError( hipDeviceSynchronize() );

    launch_divergence(dx, dy, dz,
                      dxdx, dydy, dzdz,
                      width, height, depth,
                      block_dimension,
                      grid_dimension,
                      grid_dimension_x,
                      grid_dimension_y,
                      grid_dimension_z);
    if(depth > 1)
    {
        hipFree(dzdz);
    }

    Pixel* result = new Pixel[voxel_count];
    cudaCheckError( hipMemcpy(result, dxdx, size, hipMemcpyDeviceToHost) )
    cudaCheckError( hipDeviceSynchronize() );

    hipFree(dxdx);
    hipFree(dydy);
    cudaCheckError( hipDeviceSynchronize() );

    if(is_host_data)
    {
        hipFree(dx);
        hipFree(dy);
        if(depth > 1)
            hipFree(dz);
    }

    return result;
}

template float* divergence_kernel_launch(
float* dx, float* dy, float* dz,
const uint width, const uint height, const uint depth, bool is_host_data);

template double* divergence_kernel_launch(
double* dx, double* dy, double* dz,
const uint width, const uint height, const uint depth, bool is_host_data);
