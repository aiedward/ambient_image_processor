#include "hip/hip_runtime.h"

#include "unary_operation.cu"

template<typename Pixel>
__global__ void log_kernel(
        Pixel* image,
        const uint width, const uint height, const uint depth)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height * depth)
        return;

    image[index] = log(image[index]);
}

template<typename Pixel>
Pixel* log_kernel_launch(Pixel* image_host,
                  uint width, uint height, uint depth)
{
    dim3 block_dimension;
    dim3 grid_dimension;
    Pixel* image;

    unary_operation_part1(image_host,
                      width, height, depth,
                      &image,
                      block_dimension, grid_dimension);

    log_kernel<<<grid_dimension, block_dimension>>>(image, width, height, depth);
    cudaCheckError( hipDeviceSynchronize() );

    return unary_operation_part2(image, width, height, depth);
}

template float* log_kernel_launch(float* image,
                  uint width, uint height, uint depth);
template double* log_kernel_launch(double* image,
                  uint width, uint height, uint depth);
