#include "hip/hip_runtime.h"

#include "cuda_helper.cuh"

template<typename Pixel>
__global__  void cosine_transform_kernel_3D(Pixel* image,
                              uint width, uint height, uint depth,
                              Pixel* result)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height * depth)
        return;

    const int z = floorf(index / (width*height));
    int index_rest = index - z * (width*height);
    const int y = floorf(index_rest / width);
    index_rest = index_rest - y * width;
    const int x = index_rest;

    const Pixel widthf = width;
    const Pixel heightf = height;
    const Pixel depthf = depth;

    result[index] = 0;
    for(uint z2 = 0; z2 < depth; z2++)
    {
        for(uint y2 = 0; y2 < height; y2++)
        {
            for(uint x2 = 0; x2 < width; x2++)
            {
                uint index2 = z2 * width*height + x2 + y2 * width;
                result[index] += image[index2]
                        * cospi((x2 + 0.5) * x/widthf)
                        * cospi((y2 + 0.5) * y/heightf)
                        * cospi((z2 + 0.5) * z/depthf);
            }
        }
    }
}

template<typename Pixel>
__global__  void cosine_transform_kernel_2D(Pixel* image,
                              uint width, uint height,
                              Pixel* result)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height)
        return;

    const int y = floorf(index / width);
    const int x = index - y * width;

    const Pixel widthf = width;
    const Pixel heightf = height;

    result[index] = 0;
    for(uint y2 = 0; y2 < height; y2++)
    {
        for(uint x2 = 0; x2 < width; x2++)
        {
            uint index2 = x2 + y2 * width;
            result[index] += image[index2]
                    * cospi((x2 + 0.5) * x/widthf)
                    * cospi((y2 + 0.5) * y/heightf);
        }
    }
}

template<typename Pixel>
__global__  void cosine_transform_kernel_x_2D(Pixel* image,
                              uint width, uint height,
                              Pixel* result)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height)
        return;

    const int y = floorf(index / width);
    const int x = index - y * width;

    const Pixel size = width;

    result[index] = 0;
    for(uint x2 = 0; x2 < width; x2++)
    {
        uint index2 = x2 + y * width;
        result[index] += image[index2]
                * cospi((x2 + 0.5) * x/size);
    }
 //   result[index] *= 2;
}

template<typename Pixel>
__global__  void cosine_transform_kernel_y_2D(Pixel* image,
                              uint width, uint height,
                              Pixel* result)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height)
        return;

    const int y = floorf(index / width);
    const int x = index - y * width;

    const Pixel size = height;

    result[index] = 0;
    for(uint y2 = 0; y2 < height; y2++)
    {
        uint index2 = x + y2 * width;
        result[index] += image[index2]
                * cospi((y2 + 0.5) * y/size);
    }
    result[index] *= 8;
}

template<typename Pixel>
__global__  void cosine_transform_kernel_x_3D(Pixel* image,
                              uint width, uint height, uint depth,
                              Pixel* result)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height * depth)
        return;

    const int z = floorf(index / (width*height));
    int index_rest = index - z * (width*height);
    const int y = floorf(index_rest / width);
    index_rest = index_rest - y * width;
    const int x = index_rest;

    const Pixel size = width;

    result[index] = 0;
    for(uint x2 = 0; x2 < width; x2++)
    {
        uint index2 = z * width*height + x2 + y * width;
        result[index] += image[index2]
                * cospi((x2 + 0.5) * x/size);
    }
 //   result[index] *= 2;
}

template<typename Pixel>
__global__  void cosine_transform_kernel_y_3D(Pixel* image,
                              uint width, uint height, uint depth,
                              Pixel* result)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height * depth)
        return;

    const int z = floorf(index / (width*height));
    int index_rest = index - z * (width*height);
    const int y = floorf(index_rest / width);
    index_rest = index_rest - y * width;
    const int x = index_rest;

    const Pixel size = height;

    result[index] = 0;
    for(uint y2 = 0; y2 < height; y2++)
    {
        uint index2 = z * width*height + x + y2 * width;
        result[index] += image[index2]
                * cospi((y2 + 0.5) * y/size);
    }
 //   result[index] *= 2;
}

template<typename Pixel>
__global__  void cosine_transform_kernel_z_3D(Pixel* image,
                              uint width, uint height, uint depth,
                              Pixel* result)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height * depth)
        return;

    const int z = floorf(index / (width*height));
    int index_rest = index - z * (width*height);
    const int y = floorf(index_rest / width);
    index_rest = index_rest - y * width;
    const int x = index_rest;

    const Pixel size = depth;

    result[index] = 0;
    for(uint z2 = 0; z2 < depth; z2++)
    {
        uint index2 = z2 * width*height + x + y * width;
        result[index] += image[index2]
                * cospi((z2 + 0.5) * z/size);
    }
    result[index] *= 8;
}

template<typename Pixel>
Pixel* cosine_transform_kernel_launch(Pixel* image_host,
                              uint width, uint height, uint depth)
{
    int cuda_device_count;
    cudaCheckError( hipGetDeviceCount(&cuda_device_count) );

//    printf("found %d cuda devices.\n", cuda_device_count);

    uint voxel_count = width*height*depth;
    dim3 block_dimension(CUDA_BLOCK_DIMENSON);
    dim3 grid_dimension((voxel_count + block_dimension.x - 1) / block_dimension.x);
    dim3 grid_dimension_x((width + block_dimension.x - 1) / block_dimension.x);
    dim3 grid_dimension_y((height + block_dimension.x - 1) / block_dimension.x);

    Pixel* image, *result;

    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMallocManaged(&image, size) )
    cudaCheckError( hipMemcpy(image, image_host, size, hipMemcpyHostToDevice) )

    cudaCheckError( hipMallocManaged(&result, size) )
    cudaCheckError( hipDeviceSynchronize() );

    if(depth == 1)
    {
      //  cosine_transform_kernel_2D<<<grid_dimension, block_dimension>>>(
      //    image, width, height, result);

        // separable product
        cosine_transform_kernel_x_2D<<<grid_dimension, block_dimension>>>(
          image, width, height,
          result);
        cosine_transform_kernel_y_2D<<<grid_dimension, block_dimension>>>(
          result, width, height,
          image);
        // swap
        Pixel* temp = result;
        result = image;
        image = temp;
    }
    else
    {
     //   cosine_transform_kernel_3D<<<grid_dimension, block_dimension>>>(
     //     image, width, height, depth,
      //    result);

        // separable product
        cosine_transform_kernel_x_3D<<<grid_dimension, block_dimension>>>(
          image, width, height, depth,
          result);
        cosine_transform_kernel_y_3D<<<grid_dimension, block_dimension>>>(
          result, width, height, depth,
          image);
        cosine_transform_kernel_z_3D<<<grid_dimension, block_dimension>>>(
          image, width, height, depth,
          result);
    }
    cudaCheckError( hipDeviceSynchronize() );

    Pixel* result_host = new Pixel[voxel_count];
    cudaCheckError( hipMemcpy(result_host, result, size, hipMemcpyDeviceToHost) );
    cudaCheckError( hipDeviceSynchronize() );

    hipFree(image);
    hipFree(result);

    return result_host;
}

template float* cosine_transform_kernel_launch(float* image_host,
                              uint width, uint height, uint depth);
template double* cosine_transform_kernel_launch(double* image_host,
                              uint width, uint height, uint depth);
