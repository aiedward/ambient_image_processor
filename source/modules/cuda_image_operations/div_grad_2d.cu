/*
    Ambient Image Processor - A tool to perform several imaging tasks
    
    Copyright (C) 2016 Josef Koller

    https://github.com/josefkoller/ambient_image_processor    
    
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "unary_operation.cu"
#include "tgv_common_2d.cu"

template<typename Pixel>
void launch_divergence_2d(
        Pixel* dx, Pixel* dy,
        Pixel* dxdx, Pixel* dydy,

        const uint width, const uint height,

        dim3 block_dimension,
        dim3 grid_dimension,
        dim3 grid_dimension_x,
        dim3 grid_dimension_y);

template<typename Pixel>
void tgv_launch_forward_differences_2d(Pixel* u_bar,
        Pixel* p_x, Pixel* p_y,
        uint width, uint height,
        dim3 block_dimension,
        dim3 grid_dimension_x,
        dim3 grid_dimension_y);

template<typename Pixel>
Pixel* div_grad_2d_kernel_launch(Pixel* image_host,
                  uint width, uint height)
{
    dim3 block_dimension;
    dim3 grid_dimension;
    Pixel* image;

    const uint depth = 1;
    unary_operation_part1(image_host,
                      width, height, depth,
                      &image,
                      block_dimension, grid_dimension);

    Pixel* grad_x, *grad_y;
    Pixel *dgrad_y;
    uint voxel_count = width*height*depth;
    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMalloc(&grad_x, size) )
    cudaCheckError( hipMalloc(&grad_y, size) )

    cudaCheckError( hipMalloc(&dgrad_y, size) )

    dim3 grid_dimension_x = dim3((depth*height + block_dimension.x - 1) / block_dimension.x);
    dim3 grid_dimension_y = dim3((depth*width + block_dimension.x - 1) / block_dimension.x);

    tgv_launch_forward_differences_2d(image, grad_x, grad_y, width, height,
                                      block_dimension, grid_dimension_x, grid_dimension_y);

    launch_divergence_2d(grad_x, grad_y,
                      image, dgrad_y,
                      width, height,
                      block_dimension,
                      grid_dimension,
                      grid_dimension_x,
                      grid_dimension_y);

    hipFree(grad_x);
    hipFree(grad_y);
    hipFree(dgrad_y);

    return unary_operation_part2(image, width, height, depth);
}

template float* div_grad_2d_kernel_launch(float* image,
                  uint width, uint height);
template double* div_grad_2d_kernel_launch(double* image,
                  uint width, uint heighth);

