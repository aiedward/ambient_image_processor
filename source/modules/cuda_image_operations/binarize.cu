#include "hip/hip_runtime.h"

#include "unary_operation.cu"

template<typename Pixel>
__global__ void binarize_kernel(
        Pixel* image,
        const uint width, const uint height, const uint depth)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height * depth)
        return;

    image[index] = abs(image[index]) > 1e-7 ? 1.0 : 0.0;
}

template<typename Pixel>
Pixel* binarize_kernel_launch(Pixel* image_host,
                  uint width, uint height, uint depth)
{
    dim3 block_dimension;
    dim3 grid_dimension;
    Pixel* image;

    unary_operation_part1(image_host,
                      width, height, depth,
                      &image,
                      block_dimension, grid_dimension);

    binarize_kernel<<<grid_dimension, block_dimension>>>(image, width, height, depth);

    return unary_operation_part2(image, width, height, depth);
}

template float* binarize_kernel_launch(float* image,
                  uint width, uint height, uint depth);
template double* binarize_kernel_launch(double* image,
                  uint width, uint height, uint depth);
