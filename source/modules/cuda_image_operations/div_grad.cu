
#include "unary_operation.cu"
#include "tgv_common.cu"


template<typename Pixel>
void launch_divergence(
        Pixel* dx, Pixel* dy, Pixel* dz,
        Pixel* dxdx, Pixel* dydy, Pixel* dzdz,

        const uint width, const uint height, const uint depth,

        dim3 block_dimension,
        dim3 grid_dimension,
        dim3 grid_dimension_x,
        dim3 grid_dimension_y,
        dim3 grid_dimension_z);

template<typename Pixel>
Pixel* div_grad_kernel_launch(Pixel* image_host,
                  uint width, uint height, uint depth)
{
    dim3 block_dimension;
    dim3 grid_dimension;
    Pixel* image;

    unary_operation_part1(image_host,
                      width, height, depth,
                      &image,
                      block_dimension, grid_dimension);

    Pixel* grad_x, *grad_y, *grad_z;
    Pixel *dgrad_y, *dgrad_z;
    uint voxel_count = width*height*depth;
    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMalloc(&grad_x, size) )
    cudaCheckError( hipMalloc(&grad_y, size) )
    cudaCheckError( hipMalloc(&grad_z, size) )
    cudaCheckError( hipMalloc(&dgrad_y, size) )
    cudaCheckError( hipMalloc(&dgrad_z, size) )

    dim3 grid_dimension_x = dim3((depth*height + block_dimension.x - 1) / block_dimension.x);
    dim3 grid_dimension_y = dim3((depth*width + block_dimension.x - 1) / block_dimension.x);
    dim3 grid_dimension_z = dim3((width*height + block_dimension.x - 1) / block_dimension.x);

    tgv_launch_forward_differences(image, grad_x, grad_y, grad_z,
                                   width, height, depth,
                                   block_dimension,
                                   grid_dimension_x, grid_dimension_y, grid_dimension_z);

    launch_divergence(grad_x, grad_y, grad_z,
                      image, dgrad_y, dgrad_z,
                      width, height, depth,
                      block_dimension,
                      grid_dimension,
                      grid_dimension_x,
                      grid_dimension_y,
                      grid_dimension_z);

    hipFree(grad_x);
    hipFree(grad_y);
    hipFree(grad_z);
    hipFree(dgrad_y);
    hipFree(dgrad_z);

    return unary_operation_part2(image, width, height, depth);
}

template float* div_grad_kernel_launch(float* image,
                  uint width, uint height, uint depth);
template double* div_grad_kernel_launch(double* image,
                  uint width, uint height, uint depth);

