#include "hip/hip_runtime.h"

#include "cuda_helper.cuh"

template<typename Pixel>
__global__  void convolution3x3_kernel(Pixel* image,
                              uint width, uint height,
                              Pixel* kernel, Pixel* result)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height)
        return;

    const int y = floorf(index / width);
    const int x = index - y * width;

    result[index] = image[index] * kernel[4];

    bool is_not_left = x > 0;
    bool is_not_top = y > 0;
    bool is_not_bottom = y < height - 1;
    bool is_not_right = x < width - 1;

    if(is_not_left)
    {
        result[index] += image[index-1] * kernel[3];
        if(is_not_top)
            result[index] += image[index-1-width] * kernel[0];
        if(is_not_bottom)
            result[index] += image[index-1+width] * kernel[6];
    }
    if(is_not_top)
        result[index] += image[index-width] * kernel[1];
    if(is_not_bottom)
        result[index] += image[index+width] * kernel[7];

    if(is_not_right)
    {
        result[index] += image[index+1] * kernel[5];
        if(is_not_top)
            result[index] += image[index+1-width] * kernel[2];
        if(is_not_bottom)
            result[index] += image[index+1+width] * kernel[8];
    }
}

template<typename Pixel>
__global__  void convolution3x3_with_dynamic_center_kernel(Pixel* image,
                              uint width, uint height,
                              Pixel* kernel, Pixel* result)
{
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index >= width * height)
        return;

    const int y = floorf(index / width);
    const int x = index - y * width;

    result[index] = 0;

    bool is_not_left = x > 0;
    bool is_not_top = y > 0;
    bool is_not_bottom = y < height - 1;
    bool is_not_right = x < width - 1;

    Pixel center_sum = 0;

    if(is_not_left)
    {
        result[index] += image[index-1] * kernel[3]; center_sum+= kernel[3];
        if(is_not_top) {
            result[index] += image[index-1-width] * kernel[0]; center_sum+= kernel[0];
        }
        if(is_not_bottom) {
            result[index] += image[index-1+width] * kernel[6]; center_sum+= kernel[6];
        }
    }
    if(is_not_top) {
        result[index] += image[index-width] * kernel[1]; center_sum+= kernel[1];
    }
    if(is_not_bottom) {
        result[index] += image[index+width] * kernel[7]; center_sum+= kernel[7];
    }

    if(is_not_right)
    {
        result[index] += image[index+1] * kernel[5]; center_sum+= kernel[5];
        if(is_not_top) {
            result[index] += image[index+1-width] * kernel[2]; center_sum+= kernel[2];
        }
        if(is_not_bottom) {
            result[index] += image[index+1+width] * kernel[8]; center_sum+= kernel[8];
        }
    }

    result[index] -= image[index] * center_sum;
}


template<typename Pixel>
Pixel* convolution3x3_kernel_launch(Pixel* image_host,
                              uint width, uint height,
                                    Pixel* kernel_host, bool calculate_center_as_sum_of_others)
{
    int cuda_device_count;
    cudaCheckError( hipGetDeviceCount(&cuda_device_count) );

//    printf("found %d cuda devices.\n", cuda_device_count);

    uint voxel_count = width*height;
    dim3 block_dimension(CUDA_BLOCK_DIMENSON);
    dim3 grid_dimension((voxel_count + block_dimension.x - 1) / block_dimension.x);

    Pixel* image, *kernel, *result;

    size_t size = sizeof(Pixel) * voxel_count;
    cudaCheckError( hipMalloc(&image, size) )
    cudaCheckError( hipMemcpy(image, image_host, size, hipMemcpyHostToDevice) )

    size_t kernel_size = sizeof(Pixel) * 9;
    cudaCheckError( hipMalloc(&kernel, size) )
    cudaCheckError( hipMemcpy(kernel, kernel_host, kernel_size, hipMemcpyHostToDevice) )

    cudaCheckError( hipMalloc(&result, size) )

    cudaCheckError( hipDeviceSynchronize() );

    if(!calculate_center_as_sum_of_others)
        convolution3x3_kernel<<<grid_dimension, block_dimension>>>(
          image, width, height,
          kernel, result);
    else
        convolution3x3_with_dynamic_center_kernel<<<grid_dimension, block_dimension>>>(
          image, width, height,
          kernel, result);
    cudaCheckError( hipDeviceSynchronize() );

    Pixel* result_host = new Pixel[voxel_count];
    cudaCheckError( hipMemcpy(result_host, result, size, hipMemcpyDeviceToHost) );
    cudaCheckError( hipDeviceSynchronize() );

    hipFree(image);
    hipFree(kernel);
    hipFree(result);

    return result_host;
}

template float* convolution3x3_kernel_launch(float* image_host,
                              uint width, uint height,
                              float* kernel_host, bool calculate_center_as_sum_of_others);
template double* convolution3x3_kernel_launch(double* image_host,
                              uint width, uint height,
                              double* kernel_host, bool calculate_center_as_sum_of_others);
