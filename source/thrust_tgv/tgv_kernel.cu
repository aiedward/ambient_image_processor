#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/adjacent_difference.h>
#include <algorithm>
#include <hip/hip_runtime.h>

#include "Image.cuh"
#include "thrust_operators.cuh"

__host__ __device__
Image* filter(Image* f,
    const float lambda,
    const unsigned int iteration_count)
{
#if THRUST_HOST_SYSTEM == THRUST_HOST_SYSTEM_OMP
  omp_set_num_threads(NUMBER_OF_THREADS);
#endif

  const float sqrt_8 = std::sqrt(8.0f); // algorithm paramteter
  float tau = 1.0f / sqrt_8;
  float sigma = tau;
//  const float gamma = 0.7f * lambda;  // algorithm paramteter
  float theta = 1.0f; // will be used later

  Image* u = f->clone();
  Image* p_x_temp = u->clone_uninitialized(); // memory allocation
  Image* p_y_temp = u->clone_uninitialized(); // memory allocation
  Image* p_x = u->clone_initialized(0); // memory allocation
  Image* p_y = u->clone_initialized(0); // memory allocation
  Image* p_xx_temp = u->clone_uninitialized(); // memory allocation
  Image* p_yy_temp = u->clone_uninitialized(); // memory allocation
  Image* divergence_p = u->clone_uninitialized(); // memory allocation
  Image* u_bar = u->clone(); // memory allocation
  Image* u_previous = u->clone_uninitialized(); // memory allocation

  for(uint iteration_index = 0; iteration_index < iteration_count; iteration_index++)
  {
      u_previous->setPixelDataOf(u);

      u_bar->forward_difference_x(p_x_temp);
      u_bar->forward_difference_y(p_y_temp);

      thrust::transform(p_x_temp->pixel_rows.begin(), p_x_temp->pixel_rows.end(),
                        p_x->pixel_rows.begin(), p_x_temp->pixel_rows.begin(),
                        MultiplyByConstantAndAddOperation<Pixel>(sigma) );
      thrust::transform(p_y_temp->pixel_rows.begin(), p_y_temp->pixel_rows.end(),
                        p_y->pixel_rows.begin(), p_y_temp->pixel_rows.begin(),
                        MultiplyByConstantAndAddOperation<Pixel>(sigma) );

      Image::projected_gradient(p_x_temp, p_y_temp, p_x, p_y);

      Image::divergence(p_x, p_y, p_xx_temp, p_yy_temp, divergence_p);
      thrust::transform(divergence_p->pixel_rows.begin(), divergence_p->pixel_rows.end(),
                        u->pixel_rows.begin(), u->pixel_rows.begin(),
                        MultiplyByConstantAndAddOperation<Pixel>(-tau) ); // minus goes here

      thrust::transform(f->pixel_rows.begin(), f->pixel_rows.end(),
                        u->pixel_rows.begin(), u->pixel_rows.begin(),
                        MultiplyByConstantAndAddOperation<Pixel>(tau*lambda) );

      u->scale(1/(1 + tau*lambda), u);

      thrust::transform(u->pixel_rows.begin(), u->pixel_rows.end(),
                        u_previous->pixel_rows.begin(), u_bar->pixel_rows.begin(),
                        MultiplyByConstantAndAddOperation<Pixel>(-theta));
      u_bar->add(u, u_bar);

      std::cout << "iteration: " << (iteration_index+1) << "/" << iteration_count << std::endl;
  }

  delete p_x_temp;
  delete p_y_temp;
  delete p_x;
  delete p_y;
  delete p_xx_temp;
  delete p_yy_temp;
  delete divergence_p;
  delete u_previous;
  delete u_bar;

  return u;
}


