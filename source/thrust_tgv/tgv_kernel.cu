#include "hip/hip_runtime.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/adjacent_difference.h>
#include <algorithm>
#include <hip/hip_runtime.h>

#include "Image.cuh"
#include "thrust_operators.cuh"

__host__ __device__
Image* filter(Image* f,
    const float lambda,
    const unsigned int iteration_count)
{
#if THRUST_HOST_SYSTEM == THRUST_HOST_SYSTEM_OMP
  omp_set_num_threads(NUMBER_OF_THREADS);
#endif

  const float sqrt_8 = std::sqrt(8.0f); // algorithm paramteter
  float tau = 1.0f / sqrt_8;
  float sigma = tau;
  const float gamma = 0.7f * lambda;  // algorithm paramteter
  float theta = 0.0f; // will be used later


  Image* laplace_f = f->clone_uninitialized();
  f->laplace(laplace_f);

  Image* gradient_x_f = f->clone_uninitialized();
  f->forward_difference_x(gradient_x_f);
  Image* gradient_y_f = f->clone_uninitialized();
  f->forward_difference_y(gradient_y_f);

  Image* u = f;
  Image* p_x_temp = u->clone_uninitialized(); // memory allocation
  Image* p_y_temp = u->clone_uninitialized(); // memory allocation
  Image* p_x = u->clone_initialized(0); // memory allocation
  Image* p_y = u->clone_initialized(0); // memory allocation
  Image* p_xx_temp = u->clone_uninitialized(); // memory allocation
  Image* p_yy_temp = u->clone_uninitialized(); // memory allocation
  Image* divergence_p = u->clone_uninitialized(); // memory allocation
  Image* u_step = u->clone_uninitialized(); // memory allocation
  Image* u_candidate = u->clone_uninitialized(); // memory allocation

  Image* gradient_magnitude_u = u->clone_uninitialized();
  Image* gradient_x_difference = u->clone_uninitialized();
  Image* gradient_y_difference = u->clone_uninitialized();
  Image* gradient_difference_magnitude = u->clone_uninitialized();
  for(uint iteration_index = 0; iteration_index < iteration_count; iteration_index++)
  {
      u->forward_difference_x(p_x_temp);
      u->forward_difference_y(p_y_temp);

      // begin computing energy...
      /* energy, matlab:
       temp_p = nabla * u;
       gap = sum(sqrt(temp_p(1:N).^2 + temp_p(N+1:2*N).^2)) +...
           sum((nabla * u - nabla * f).^2) * lambda/2;
       */

      /*
      thrust::transform(p_x_temp->pixel_rows.begin(), p_x_temp->pixel_rows.end(),
                        p_y_temp->pixel_rows.begin(), gradient_magnitude_u->pixel_rows.begin(),
                        GradientMagnitude<Pixel>());
      float energy1 = thrust::reduce(gradient_magnitude_u->pixel_rows.begin(),
                               gradient_magnitude_u->pixel_rows.end(),
                               0, // init
                               thrust::plus<Pixel>() );

      thrust::transform(p_x_temp->pixel_rows.begin(), p_x_temp->pixel_rows.end(),
                        gradient_x_f->pixel_rows.begin(),
                        gradient_x_difference->pixel_rows.begin(),
                        thrust::minus<Pixel>());
      thrust::transform(p_y_temp->pixel_rows.begin(), p_y_temp->pixel_rows.end(),
                        gradient_y_f->pixel_rows.begin(),
                        gradient_y_difference->pixel_rows.begin(),
                        thrust::minus<Pixel>());
      thrust::transform(gradient_x_difference->pixel_rows.begin(), gradient_x_difference->pixel_rows.end(),
                        gradient_y_difference->pixel_rows.begin(), gradient_difference_magnitude->pixel_rows.begin(),
                        GradientMagnitudeSquare<Pixel>());
      std::cout << "energy1: " << energy1 << std::endl;
      float energy = energy1 + lambda*0.5f * thrust::reduce(
                  gradient_difference_magnitude->pixel_rows.begin(),
                  gradient_difference_magnitude->pixel_rows.end(),
                  0, // init
                  thrust::plus<Pixel>());
      std::cout << "energy: " << energy << std::endl;
      // end energy
      */

      // p update ... matlab: temp_p  = nabla * head_u * sigma + p;
      thrust::transform(p_x_temp->pixel_rows.begin(), p_x_temp->pixel_rows.end(),
                        p_x->pixel_rows.begin(), p_x_temp->pixel_rows.begin(),
                        MultiplyByConstantAndAddOperation<Pixel>(sigma) );
      thrust::transform(p_y_temp->pixel_rows.begin(), p_y_temp->pixel_rows.end(),
                        p_y->pixel_rows.begin(), p_y_temp->pixel_rows.begin(),
                        MultiplyByConstantAndAddOperation<Pixel>(sigma) );// JJ reuse operation?
      // project/normalize p
      Image::projected_gradient(p_x_temp, p_y_temp, p_x, p_y);

      // u update1...  matlab: u = old_u + tau * lambda * (-nabla_t * p * (1/lambda + 1)+ laplace_f);
      Image::divergence(p_x, p_y, p_xx_temp, p_yy_temp, divergence_p);

      thrust::transform(divergence_p->pixel_rows.begin(), divergence_p->pixel_rows.end(),
                        laplace_f->pixel_rows.begin(), u_candidate->pixel_rows.begin(),
                        MultiplyByConstantAndAddOperation<Pixel>(-(1.0f/lambda + 1.0f)) ); // minus goes here
      thrust::transform(u_candidate->pixel_rows.begin(), u_candidate->pixel_rows.end(),
                        u->pixel_rows.begin(), u_candidate->pixel_rows.begin(),
                        MultiplyByConstantAndAddOperation<Pixel>(tau*lambda));

      // step sizes update
      theta = 1.0f / std::sqrt(1.0f + 2.0f * gamma * tau);
      tau *= theta;
      sigma /= theta;

      thrust::transform(u_candidate->pixel_rows.begin(), u_candidate->pixel_rows.end(),
                        u->pixel_rows.begin(), u_step->pixel_rows.begin(),
                        thrust::minus<Pixel>());
      // u update2
      thrust::transform(u_step->pixel_rows.begin(), u_step->pixel_rows.end(),
                        u->pixel_rows.begin(), u->pixel_rows.begin(),
                        MultiplyByConstantAndAddOperation<Pixel>(theta) );

      std::cout << "iteration: " << (iteration_index+1) << "/" << iteration_count << std::endl;
  }
  delete gradient_difference_magnitude;
  delete gradient_x_difference;
  delete gradient_y_difference;
  delete gradient_x_f;
  delete gradient_y_f;
  delete gradient_magnitude_u;

  delete laplace_f;
  delete p_x_temp;
  delete p_y_temp;
  delete p_x;
  delete p_y;
  delete p_xx_temp;
  delete p_yy_temp;
  delete divergence_p;
  delete u_step;
  delete u_candidate;

  return u;
}


